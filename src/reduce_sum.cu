#include <hipcub/hipcub.hpp>
#include <hip/hip_runtime_api.h>

namespace duda
{

namespace detail
{

template <typename T>
inline T reduce_sum(const T* const data, const int size)
{
    T* out_d          = NULL;
    void* tmp_storage = NULL;

    const auto code1 = hipMalloc((void**)&out_d, sizeof(T));

    std::size_t tmp_storage_bytes = 0;

    const auto code2 = hipcub::DeviceReduce::Sum(
        tmp_storage, tmp_storage_bytes, data, out_d, size);

    const auto code3 = hipMalloc(&tmp_storage, tmp_storage_bytes);

    const auto code4 = hipcub::DeviceReduce::Sum(
        tmp_storage, tmp_storage_bytes, data, out_d, size);

    T* out_h = new T[1];

    hipMemcpy(out_h, out_d, sizeof(T) * 1, hipMemcpyDeviceToHost);

    const T out = out_h[0];

    const auto code5 = hipFree(tmp_storage);

    const auto code6 = hipFree(out_d);

    delete[] out_h;

    return out;
}

} // namespace detail

float reduce_sum(const float* const data, const int size)
{
    return detail::reduce_sum(data, size);
}

double reduce_sum(const double* const data, const int size)
{
    return detail::reduce_sum(data, size);
}

} // namespace duda
