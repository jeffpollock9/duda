#include "reduce_sum.hpp"

#include "check_error.hpp"

#include <hipcub/hipcub.hpp>
#include <hip/hip_runtime_api.h>

namespace duda
{

namespace detail
{

template <typename T>
inline T reduce_sum(const T* const data, const int size)
{
    using dr = hipcub::DeviceReduce;

    T* out_d          = NULL;
    void* tmp_storage = NULL;

    check_cuda_error(hipMalloc((void**)&out_d, sizeof(T)));

    std::size_t tmp_storage_bytes = 0;

    check_cuda_error(
        dr::Sum(tmp_storage, tmp_storage_bytes, data, out_d, size));

    check_cuda_error(hipMalloc(&tmp_storage, tmp_storage_bytes));

    check_cuda_error(
        dr::Sum(tmp_storage, tmp_storage_bytes, data, out_d, size));

    check_cuda_error(hipFree(tmp_storage));

    T out;

    check_cuda_error(
        hipMemcpy(&out, out_d, sizeof(T), hipMemcpyDeviceToHost));

    check_cuda_error(hipFree(out_d));

    return out;
}

} // namespace detail

float reduce_sum(const float* const data, const int size)
{
    return detail::reduce_sum(data, size);
}

double reduce_sum(const double* const data, const int size)
{
    return detail::reduce_sum(data, size);
}

} // namespace duda
