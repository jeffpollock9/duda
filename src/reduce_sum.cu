#include <duda/reduce_sum.hpp>
#include <duda/check_error.hpp>

#include <hipcub/hipcub.hpp>
#include <hip/hip_runtime_api.h>

namespace duda
{

namespace detail
{

template <typename T>
inline T reduce_sum(const T* const data, const int size)
{
    using dr = hipcub::DeviceReduce;

    T* out_d          = nullptr;
    void* tmp_storage = nullptr;

    check_error(hipMalloc((void**)&out_d, sizeof(T)));

    std::size_t tmp_storage_bytes = 0;

    check_error(dr::Sum(tmp_storage, tmp_storage_bytes, data, out_d, size));

    check_error(hipMalloc(&tmp_storage, tmp_storage_bytes));

    check_error(dr::Sum(tmp_storage, tmp_storage_bytes, data, out_d, size));

    check_error(hipFree(tmp_storage));

    T out;

    check_error(hipMemcpy(&out, out_d, sizeof(T), hipMemcpyDeviceToHost));

    check_error(hipFree(out_d));

    return out;
}

} // namespace detail

int reduce_sum(const int* const data, const int size)
{
    return detail::reduce_sum(data, size);
}

float reduce_sum(const float* const data, const int size)
{
    return detail::reduce_sum(data, size);
}

double reduce_sum(const double* const data, const int size)
{
    return detail::reduce_sum(data, size);
}

} // namespace duda
