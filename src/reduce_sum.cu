#include <hipcub/hipcub.hpp>
#include <hip/hip_runtime_api.h>

namespace duda
{

namespace detail
{

template <typename T>
inline T reduce_sum(T* const data, const int size)
{
    // storage for answer
    T* out_d = NULL;

    const hipError_t code1 = hipMalloc((void**)&out_d, sizeof(T) * 1);

    // get temp storage
    void* tmp_storage = NULL;

    std::size_t tmp_storage_bytes = 0;

    const hipError_t code2 = hipcub::DeviceReduce::Sum(
        tmp_storage, tmp_storage_bytes, data, out_d, size);

    const hipError_t code3 = hipMalloc(&tmp_storage, tmp_storage_bytes);

    // do the reduction
    const hipError_t code4 = hipcub::DeviceReduce::Sum(
        tmp_storage, tmp_storage_bytes, data, out_d, size);

    // copy answer to host
    T* out_h = new T[1];

    hipMemcpy(out_h, out_d, sizeof(T) * 1, hipMemcpyDeviceToHost);

    const T out = out_h[0];

    // tidy up
    const hipError_t code5 = hipFree(tmp_storage);

    const hipError_t code6 = hipFree(out_d);

    delete[] out_h;

    return out;
}

} // namespace detail

float reduce_sum(float* const data, const int size)
{
    return detail::reduce_sum(data, size);
}

double reduce_sum(double* const data, const int size)
{
    return detail::reduce_sum(data, size);
}

} // namespace duda
