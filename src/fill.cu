#include "hip/hip_runtime.h"
#include "fill.hpp"

namespace duda
{

namespace kernel
{

template <typename T>
__global__ void fill(T* const data, const int size, const T value)
{
    const int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < size)
    {
        data[i] = value;
    }
}

} // namespace kernel

namespace detail
{

template <typename T>
inline void fill(T* const data, const int size, const T value)
{
    const int d = 1024;
    const int n = (size + d) / d;

    const dim3 blocks(n);
    const dim3 block_dim(d);

    kernel::fill<T><<<blocks, block_dim>>>(data, size, value);
}

} // namespace detail

void fill(float* const data, const int size, const float value)
{
    detail::fill(data, size, value);
}

void fill(double* const data, const int size, const double value)
{
    detail::fill(data, size, value);
}

} // namespace duda
